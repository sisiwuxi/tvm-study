#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 
__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;
 
    // do reduction in global mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            d_in[myId] += d_in[myId + s];
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }
 
    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = d_in[myId];
    }
}
 
__global__ void shmem_reduce_kernel(float * d_out, const float * d_in)
{
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];
 
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;
 
    // load shared mem from global mem
    sdata[tid] = d_in[myId];
    __syncthreads();            // make sure entire block is loaded!
 
    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }
 
    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}
 
void reduce(float * d_out, float * d_intermediate, float * d_in, 
            int size, bool usesSharedMemory)
{
    // assumes that size is not greater than maxThreadsPerBlock^2
    // and that size is a multiple of maxThreadsPerBlock
    const int maxThreadsPerBlock = 1024;
    int threads = maxThreadsPerBlock;
    int blocks = size / maxThreadsPerBlock;
    if (usesSharedMemory)
    {
        shmem_reduce_kernel<<<blocks, threads, threads * sizeof(float)>>>
            (d_intermediate, d_in);
    }
    else
    {
        global_reduce_kernel<<<blocks, threads>>>
            (d_intermediate, d_in);
    }
    // now we're down to one block left, so reduce it
    threads = blocks; // launch one thread for each block in prev step
    blocks = 1;
    if (usesSharedMemory)
    {
        shmem_reduce_kernel<<<blocks, threads, threads * sizeof(float)>>>
            (d_out, d_intermediate);
    }
    else
    {
        global_reduce_kernel<<<blocks, threads>>>
            (d_out, d_intermediate);
    }
}
 
int main(int argc, char **argv)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    int dev = 0;
    hipSetDevice(dev);
 
    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
               devProps.name, (int)devProps.totalGlobalMem, 
               (int)devProps.major, (int)devProps.minor, 
               (int)devProps.clockRate);
    }
 
    const int ARRAY_SIZE = 1 << 20;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
 
    // generate the input array on the host
    float h_in[ARRAY_SIZE];
    float sum = 0.0f;
    srand(0);
    for(int i = 0; i < ARRAY_SIZE; i++) {
        // generate random float in [-1.0f, 1.0f]
        h_in[i] = -1.0f + (float)random()/((float)RAND_MAX/2.0f);
        sum += h_in[i];
    }
 
    // declare GPU memory pointers
    float * d_in, * d_intermediate, * d_out;
 
    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_intermediate, ARRAY_BYTES); // overallocated
    hipMalloc((void **) &d_out, sizeof(float));
 
    // transfer the input array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice); 
 
    int whichKernel = 0;
    if (argc == 2) {
        whichKernel = atoi(argv[1]);
    }
        
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // launch the kernel
    switch(whichKernel) {
    case 0:
        printf("Running global reduce\n");
        hipEventRecord(start, 0);
        for (int i = 0; i < 100; i++)
        {
            reduce(d_out, d_intermediate, d_in, ARRAY_SIZE, false);
        }
        hipEventRecord(stop, 0);
        break;
    case 1:
        printf("Running reduce with shared mem\n");
        hipEventRecord(start, 0);
        for (int i = 0; i < 100; i++)
        {
            reduce(d_out, d_intermediate, d_in, ARRAY_SIZE, true);
        }
        hipEventRecord(stop, 0);
        break;
    default:
        fprintf(stderr, "error: ran no kernel\n");
        exit(EXIT_FAILURE);
    }
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);    
    elapsedTime /= 100.0f;      // 100 trials
 
    // copy back the sum from GPU
    float h_out;
    hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);
 
    printf("h_out=%f, average time elapsed: %f\n", h_out, elapsedTime);
 
    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_intermediate);
    hipFree(d_out);
        
    return 0;
}