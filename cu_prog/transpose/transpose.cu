#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"

const int N= 1024;        // matrix size is NxN
const int K= 32;          // tile size is KxK

// Utility functions: compare, print, and fill matrices
#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line)
{
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA error at: %s : %d\n", file,line);
    fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);;
    exit(1);
  }
}

int compare_matrices(float *gpu, float *ref)
{
    int result = 0;

    for(int j=0; j < N; j++)
        for(int i=0; i < N; i++)
            if (ref[i + j*N] != gpu[i + j*N])
            {
                // printf("reference(%d,%d) = %f but test(%d,%d) = %f\n",
                // i,j,ref[i+j*N],i,j,test[i+j*N]);
                result = 1;
            }
    return result;
}

void print_matrix(float *mat)
{
    for(int j=0; j < N; j++) 
    {
        for(int i=0; i < N; i++) { printf("%4.4g ", mat[i + j*N]); }
        printf("\n");
    }    
}

// fill a matrix with sequential numbers in the range 0..N-1
void fill_matrix(float *mat)
{
    for(int j=0; j < N * N; j++)
        mat[j] = (float) j;
}



void 
transpose_CPU(float in[], float out[])
{
    for(int j=0; j < N; j++)
        for(int i=0; i < N; i++)
              out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched on a single thread
__global__ void 
transpose_serial(float in[], float out[])
{
    for(int j=0; j < N; j++)
        for(int i=0; i < N; i++)
            out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched with one thread per row of output matrix
__global__ void 
transpose_parallel_per_row(float in[], float out[])
{
    int i = threadIdx.x;

    for(int j=0; j < N; j++)
        out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched with one thread per element, in KxK threadblocks
// thread (x,y) in grid writes element (i,j) of output matrix 
__global__ void 
transpose_parallel_per_element(float in[], float out[])
{
    int i = blockIdx.x * K + threadIdx.x;
    int j = blockIdx.y * K + threadIdx.y;

    out[j + i*N] = in[i + j*N]; // out(j,i) = in(i,j)
}

// to be launched with one thread per element, in (tilesize)x(tilesize) threadblocks
// thread blocks read & write tiles, in coalesced fashion
// adjacent threads read adjacent input elements, write adjacent output elmts
__global__ void 
transpose_parallel_per_element_tiled(float in[], float out[])
{
    // (i,j) locations of the tile corners for input & output matrices:
    int in_corner_i  = blockIdx.x * K, in_corner_j  = blockIdx.y * K;
    int out_corner_i = blockIdx.y * K, out_corner_j = blockIdx.x * K;

    int x = threadIdx.x, y = threadIdx.y;

    __shared__ float tile[K][K];

    // coalesced read from global mem, TRANSPOSED write into shared mem:
    tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y)*N];
    __syncthreads();
    // read from shared mem, coalesced write to global mem:
    out[(out_corner_i + x) + (out_corner_j + y)*N] = tile[x][y];
}

// to be launched with one thread per element, in (tilesize)x(tilesize) threadblocks
// thread blocks read & write tiles, in coalesced fashion
// adjacent threads read adjacent input elements, write adjacent output elmts
__global__ void 
transpose_parallel_per_element_tiled16(float in[], float out[])
{
    // (i,j) locations of the tile corners for input & output matrices:
    int in_corner_i  = blockIdx.x * 16, in_corner_j  = blockIdx.y * 16;
    int out_corner_i = blockIdx.y * 16, out_corner_j = blockIdx.x * 16;

    int x = threadIdx.x, y = threadIdx.y;

    __shared__ float tile[16][16];

    // coalesced read from global mem, TRANSPOSED write into shared mem:
    tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y)*N];
    __syncthreads();
    // read from shared mem, coalesced write to global mem:
    out[(out_corner_i + x) + (out_corner_j + y)*N] = tile[x][y];
}

// to be launched with one thread per element, in KxK threadblocks
// thread blocks read & write tiles, in coalesced fashion
// shared memory array padded to avoid bank conflicts
__global__ void 
transpose_parallel_per_element_tiled_padded(float in[], float out[])
{
    // (i,j) locations of the tile corners for input & output matrices:
    int in_corner_i  = blockIdx.x * K, in_corner_j  = blockIdx.y * K;
    int out_corner_i = blockIdx.y * K, out_corner_j = blockIdx.x * K;

    int x = threadIdx.x, y = threadIdx.y;

    __shared__ float tile[K][K+1];

    // coalesced read from global mem, TRANSPOSED write into shared mem:
    tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y)*N];
    __syncthreads();
    // read from shared mem, coalesced write to global mem:
    out[(out_corner_i + x) + (out_corner_j + y)*N] = tile[x][y];
}

// to be launched with one thread per element, in KxK threadblocks
// thread blocks read & write tiles, in coalesced fashion
// shared memory array padded to avoid bank conflicts
__global__ void 
transpose_parallel_per_element_tiled_padded16(float in[], float out[])
{
    // (i,j) locations of the tile corners for input & output matrices:
    int in_corner_i  = blockIdx.x * 16, in_corner_j  = blockIdx.y * 16;
    int out_corner_i = blockIdx.y * 16, out_corner_j = blockIdx.x * 16;

    int x = threadIdx.x, y = threadIdx.y;

    __shared__ float tile[16][16+1];

    // coalesced read from global mem, TRANSPOSED write into shared mem:
    tile[y][x] = in[(in_corner_i + x) + (in_corner_j + y)*N];
    __syncthreads();
    // read from shared mem, coalesced write to global mem:
    out[(out_corner_i + x) + (out_corner_j + y)*N] = tile[x][y];
}

int main(int argc, char **argv)
{
    int numbytes = N * N * sizeof(float);

    float *in = (float *) malloc(numbytes);
    float *out = (float *) malloc(numbytes);
    float *gold = (float *) malloc(numbytes);

    fill_matrix(in);
    transpose_CPU(in, gold);

    float *d_in, *d_out;

    hipMalloc(&d_in, numbytes);
    hipMalloc(&d_out, numbytes);
    hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice);

    GpuTimer timer;

/*  
 * Now time each kernel and verify that it produces the correct result.
 *
 * To be really careful about benchmarking purposes, we should run every kernel once
 * to "warm" the system and avoid any compilation or code-caching effects, then run 
 * every kernel 10 or 100 times and average the timings to smooth out any variance. 
 * But this makes for messy code and our goal is teaching, not detailed benchmarking.
 */

    timer.Start();
    transpose_serial<<<1,1>>>(d_in, d_out);
    timer.Stop();
    hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
    printf("transpose_serial: %g ms.\nVerifying transpose...%s\n", 
           timer.Elapsed(), compare_matrices(out, gold) ? "Failed" : "Success");

    timer.Start();
    transpose_parallel_per_row<<<1,N>>>(d_in, d_out);
    timer.Stop();
    hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
    printf("transpose_parallel_per_row: %g ms.\nVerifying transpose...%s\n", 
           timer.Elapsed(), compare_matrices(out, gold) ? "Failed" : "Success");

    dim3 blocks(N/K,N/K); // blocks per grid
    dim3 threads(K,K);    // threads per block

    timer.Start();
    transpose_parallel_per_element<<<blocks,threads>>>(d_in, d_out);
    timer.Stop();
    hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
    printf("transpose_parallel_per_element: %g ms.\nVerifying transpose...%s\n",
           timer.Elapsed(), compare_matrices(out, gold) ? "Failed" : "Success");

    timer.Start();
    transpose_parallel_per_element_tiled<<<blocks,threads>>>(d_in, d_out);
    timer.Stop();
    hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
    printf("transpose_parallel_per_element_tiled %dx%d: %g ms.\nVerifying ...%s\n", 
           K, K, timer.Elapsed(), compare_matrices(out, gold) ? "Failed" : "Success");
    
    dim3 blocks16x16(N/16,N/16); // blocks per grid
    dim3 threads16x16(16,16);     // threads per block

    timer.Start();
    transpose_parallel_per_element_tiled16<<<blocks16x16,threads16x16>>>(d_in, d_out);
    timer.Stop();
    hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
    printf("transpose_parallel_per_element_tiled 16x16: %g ms.\nVerifying ...%s\n", 
           timer.Elapsed(), compare_matrices(out, gold) ? "Failed" : "Success");
    
    timer.Start();
     transpose_parallel_per_element_tiled_padded16<<<blocks16x16,threads16x16>>>(d_in, d_out);
    timer.Stop();
    hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost);
    printf("transpose_parallel_per_element_tiled_padded 16x16: %g ms.\nVerifying...%s\n", 
           timer.Elapsed(), compare_matrices(out, gold) ? "Failed" : "Success");

    hipFree(d_in);
    hipFree(d_out);
}