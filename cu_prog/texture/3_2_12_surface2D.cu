#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cu_prog.h"
// 2D surfaces
surface<void, 2> inputSurfRef;
surface<void, 2> outputSurfRef;
            
// Simple copy kernel
__global__ void copyKernel(int width, int height) 
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        uchar4 data;
        // Read from input surface
        surf2Dread(&data,  inputSurfRef, x * 4, y);
        // Write to output surface
        surf2Dwrite(data, outputSurfRef, x * 4, y);
    }
}

// Host code
int main()
{
    const int height = 32;//1024;
    const int width = 32;//1024;

    // Allocate and set some host data
    unsigned char *h_data =
        (unsigned char *)std::malloc(sizeof(unsigned char) * width * height * 4);
    for (int i = 0; i < height * width * 4; ++i)
        h_data[i] = i;
    printSurface(h_data, width, height, 4);
    // Allocate CUDA arrays in device memory
    hipChannelFormatDesc channelDesc =
             hipCreateChannelDesc(8, 8, 8, 8,
                                   hipChannelFormatKindUnsigned);
    hipArray* cuInputArray;
    hipMallocArray(&cuInputArray, &channelDesc, width, height,
                    hipArraySurfaceLoadStore);
    hipArray* cuOutputArray;
    hipMallocArray(&cuOutputArray, &channelDesc, width, height,
                    hipArraySurfaceLoadStore);

    // Copy to device memory some data located at address h_data
    // in host memory
    size_t size = 4 * width * sizeof(unsigned char) * height;
    hipMemcpyToArray(cuInputArray, 0, 0, h_data, size,hipMemcpyHostToDevice);

    // Bind the arrays to the surface references
    cudaBindSurfaceToArray(inputSurfRef, cuInputArray);
    cudaBindSurfaceToArray(outputSurfRef, cuOutputArray);

    // Invoke kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width  + dimBlock.x - 1) / dimBlock.x,
                 (height + dimBlock.y - 1) / dimBlock.y);
    copyKernel<<<dimGrid, dimBlock>>>(width, height);

    // Free device memory
    hipFreeArray(cuInputArray);
    hipFreeArray(cuOutputArray);
    // Copy data from device back to host
    hipMemcpyFromArray(h_data, cuOutputArray, 0, 0, size, hipMemcpyDeviceToHost);
    // Free device memory
    printSurface(h_data, width, height, 4);
    return 0;
}
