int getCudaDeviceForVulkanPhysicalDevice(VkPhysicalDevice vkPhysicalDevice) {
    VkPhysicalDeviceIDProperties vkPhysicalDeviceIDProperties = {};
    vkPhysicalDeviceIDProperties.sType =
    VK_STRUCTURE_TYPE_PHYSICAL_DEVICE_ID_PROPERTIES;
    vkPhysicalDeviceIDProperties.pNext = NULL;
    VkPhysicalDeviceProperties2 vkPhysicalDeviceProperties2 = {};
    vkPhysicalDeviceProperties2.sType =
    VK_STRUCTURE_TYPE_PHYSICAL_DEVICE_PROPERTIES_2;
    vkPhysicalDeviceProperties2.pNext = &vkPhysicalDeviceIDProperties;
    vkGetPhysicalDeviceProperties2(vkPhysicalDevice, &vkPhysicalDeviceProperties2);
    int cudaDeviceCount;
    hipGetDeviceCount(&cudaDeviceCount);
    for (int cudaDevice = 0; cudaDevice < cudaDeviceCount; cudaDevice++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, cudaDevice);
        if (!memcmp(&deviceProp.uuid, vkPhysicalDeviceIDProperties.deviceUUID, VK_UUID_SIZE)) {
            return cudaDevice;
        }
    }
    return hipInvalidDeviceId;
}

hipExternalMemory_t importVulkanMemoryObjectFromFileDescriptor(int fd, unsigned long long size, bool isDedicated) {
    hipExternalMemory_t extMem = NULL;
    hipExternalMemoryHandleDesc desc = {};
    memset(&desc, 0, sizeof(desc));
    desc.type = hipExternalMemoryHandleTypeOpaqueFd;
    desc.handle.fd = fd;
    desc.size = size;
    if (isDedicated) {
        desc.flags |= hipExternalMemoryDedicated;
    }
    hipImportExternalMemory(&extMem, &desc);
    // Input parameter 'fd' should not be used beyond this point as CUDA has assumed ownership of it
    return extMem;
}

hipExternalMemory_t importVulkanMemoryObjectFromNTHandle(HANDLE handle, unsigned long long size, bool isDedicated) {
    hipExternalMemory_t extMem = NULL;
    hipExternalMemoryHandleDesc desc = {};
    memset(&desc, 0, sizeof(desc));
    desc.type = hipExternalMemoryHandleTypeOpaqueWin32;
    desc.handle.win32.handle = handle;
    desc.size = size;
    if (isDedicated) {
        desc.flags |= hipExternalMemoryDedicated;
    }
    hipImportExternalMemory(&extMem, &desc);
    // Input parameter 'handle' should be closed if it's not needed anymore
    CloseHandle(handle);
    return extMem;
}

hipExternalMemory_t importVulkanMemoryObjectFromNamedNTHandle(LPCWSTR
 name, unsigned long long size, bool isDedicated) {
    hipExternalMemory_t extMem = NULL;
    hipExternalMemoryHandleDesc desc = {};
    memset(&desc, 0, sizeof(desc));
    desc.type = hipExternalMemoryHandleTypeOpaqueWin32;
    desc.handle.win32.name = (void *)name;
    desc.size = size;
    if (isDedicated) {
        desc.flags |= hipExternalMemoryDedicated;
    }
    hipImportExternalMemory(&extMem, &desc);
    return extMem;
}


hipExternalMemory_t importVulkanMemoryObjectFromKMTHandle(HANDLE handle, unsigned long long size, bool isDedicated) {
    hipExternalMemory_t extMem = NULL;
    hipExternalMemoryHandleDesc desc = {};
    memset(&desc, 0, sizeof(desc));
    desc.type = hipExternalMemoryHandleTypeOpaqueWin32Kmt;
    desc.handle.win32.handle = (void *)handle;
    desc.size = size;
    if (isDedicated) {
    desc.flags |= hipExternalMemoryDedicated;
    }
    hipImportExternalMemory(&extMem, &desc);
    return extMem;
}

void * mapBufferOntoExternalMemory(hipExternalMemory_t extMem, unsigned long long offset, unsigned long long size) {
    void *ptr = NULL;
    hipExternalMemoryBufferDesc desc = {};
    memset(&desc, 0, sizeof(desc));
    desc.offset = offset;
    desc.size = size;
    hipExternalMemoryGetMappedBuffer(&ptr, extMem, &desc);
    // Note: ‘ptr’ must eventually be freed using hipFree()

    return ptr;
}

hipMipmappedArray_t mapMipmappedArrayOntoExternalMemory(hipExternalMemory_t extMem, unsigned long long offset, hipChannelFormatDesc *formatDesc, hipExtent *extent, unsigned int flags, unsigned int numLevels) {
    hipMipmappedArray_t mipmap = NULL;
    cudaExternalMemoryMipmappedArrayDesc desc = {};
    memset(&desc, 0, sizeof(desc));
    desc.offset = offset;
    desc.formatDesc = *formatDesc;
    desc.extent = *extent;
    desc.flags = flags;
    desc.numLevels = numLevels;
    // Note: ‘mipmap’ must eventually be freed using hipFreeMipmappedArray()
    cudaExternalMemoryGetMappedMipmappedArray(&mipmap, extMem, &desc);
    return mipmap;
}
hipChannelFormatDesc getCudaChannelFormatDescForVulkanFormat(VkFormat format)
{
    hipChannelFormatDesc d;
    memset(&d, 0, sizeof(d));
    switch (format) {
        case VK_FORMAT_R8_UINT: d.x = 8; d.y = 0; d.z = 0; d.w = 0; d.f= hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R8_SINT: d.x = 8; d.y = 0; d.z = 0; d.w = 0; d.f= hipChannelFormatKindSigned; break;
        case VK_FORMAT_R8G8_UINT: d.x = 8; d.y = 8; d.z = 0; d.w = 0; d.f= hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R8G8_SINT: d.x = 8; d.y = 8; d.z = 0; d.w = 0; d.f= hipChannelFormatKindSigned; break;
        case VK_FORMAT_R8G8B8A8_UINT: d.x = 8; d.y = 8; d.z = 8; d.w = 8; d.f= hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R8G8B8A8_SINT: d.x = 8; d.y = 8; d.z = 8; d.w = 8; d.f= hipChannelFormatKindSigned; break;
        case VK_FORMAT_R16_UINT: d.x = 16; d.y = 0; d.z = 0; d.w = 0; d.f= hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R16_SINT: d.x = 16; d.y = 0; d.z = 0; d.w = 0; d.f= hipChannelFormatKindSigned; break;
        case VK_FORMAT_R16G16_UINT: d.x = 16; d.y = 16; d.z = 0; d.w = 0; d.f= hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R16G16_SINT: d.x = 16; d.y = 16; d.z = 0; d.w = 0; d.f= hipChannelFormatKindSigned; break;
        case VK_FORMAT_R16G16B16A16_UINT: d.x = 16; d.y = 16; d.z = 16; d.w = 16; d.f= hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R16G16B16A16_SINT: d.x = 16; d.y = 16; d.z = 16; d.w = 16; d.f= hipChannelFormatKindSigned; break;
        case VK_FORMAT_R32_UINT: d.x = 32; d.y = 0; d.z = 0; d.w = 0; d.f= hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R32_SINT: d.x = 32; d.y = 0; d.z = 0; d.w = 0; d.f= hipChannelFormatKindSigned; break;
        case VK_FORMAT_R32_SFLOAT: d.x = 32; d.y = 0; d.z = 0; d.w = 0; d.f= hipChannelFormatKindFloat; break;
        case VK_FORMAT_R32G32_UINT: d.x = 32; d.y = 32; d.z = 0; d.w = 0; d.f= hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R32G32_SINT: d.x = 32; d.y = 32; d.z = 0; d.w = 0; d.f= hipChannelFormatKindSigned; break;
        case VK_FORMAT_R32G32_SFLOAT: d.x = 32; d.y = 32; d.z = 0; d.w = 0; d.f= hipChannelFormatKindFloat; break;
        case VK_FORMAT_R32G32B32A32_UINT: d.x = 32; d.y = 32; d.z = 32; d.w = 32; d.f= hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R32G32B32A32_SINT: d.x = 32; d.y = 32; d.z = 32; d.w = 32; d.f= hipChannelFormatKindSigned; break;
        case VK_FORMAT_R32G32B32A32_SFLOAT: d.x = 32; d.y = 32; d.z = 32; d.w = 32; d.f= hipChannelFormatKindFloat; break;
        default: assert(0);
    }

 return d;
}

hipExtent getCudaExtentForVulkanExtent(VkExtent3D vkExt, uint32_t arrayLayers,
 VkImageViewType vkImageViewType) {
 hipExtent e = { 0, 0, 0 };
 switch (vkImageViewType) {
 case VK_IMAGE_VIEW_TYPE_1D: e.width = vkExt.width; e.height = 0;
 e.depth = 0; break;
 case VK_IMAGE_VIEW_TYPE_2D: e.width = vkExt.width; e.height =
 vkExt.height; e.depth = 0; break;
 case VK_IMAGE_VIEW_TYPE_3D: e.width = vkExt.width; e.height =
 vkExt.height; e.depth = vkExt.depth; break;
 case VK_IMAGE_VIEW_TYPE_CUBE: e.width = vkExt.width; e.height =
 vkExt.height; e.depth = arrayLayers; break;
 case VK_IMAGE_VIEW_TYPE_1D_ARRAY: e.width = vkExt.width; e.height = 0;
 e.depth = arrayLayers; break;
 case VK_IMAGE_VIEW_TYPE_2D_ARRAY: e.width = vkExt.width; e.height =
 vkExt.height; e.depth = arrayLayers; break;
 case VK_IMAGE_VIEW_TYPE_CUBE_ARRAY: e.width = vkExt.width; e.height =
 vkExt.height; e.depth = arrayLayers; break;
 default: assert(0);
 }
 return e;
}
unsigned int getCudaMipmappedArrayFlagsForVulkanImage(VkImageViewType
 vkImageViewType, VkImageUsageFlags vkImageUsageFlags, bool allowSurfaceLoadStore) {
    unsigned int flags = 0;
    switch (vkImageViewType) {
        case VK_IMAGE_VIEW_TYPE_CUBE: flags |= hipArrayCubemap;
            break;
        case VK_IMAGE_VIEW_TYPE_CUBE_ARRAY: flags |= hipArrayCubemap |hipArrayLayered; break;
        case VK_IMAGE_VIEW_TYPE_1D_ARRAY: flags |= hipArrayLayered;
            break;
        case VK_IMAGE_VIEW_TYPE_2D_ARRAY: flags |= hipArrayLayered;
            break;
        default: break;
    }
    if (vkImageUsageFlags & VK_IMAGE_USAGE_COLOR_ATTACHMENT_BIT) {
        flags |= cudaArrayColorAttachment;
    }
    if (allowSurfaceLoadStore) {
        flags |= hipArraySurfaceLoadStore;
    }
    return flags;
}


//  synchronization objects


hipExternalSemaphore_t importVulkanSemaphoreObjectFromFileDescriptor(int fd) {
    hipExternalSemaphore_t extSem = NULL;
    hipExternalSemaphoreHandleDesc desc = {};

    memset(&desc, 0, sizeof(desc));
    desc.type = hipExternalSemaphoreHandleTypeOpaqueFd;
    desc.handle.fd = fd;
    hipImportExternalSemaphore(&extSem, &desc);
    // Input parameter 'fd' should not be used beyond this point as CUDA has assumed ownership of it
    return extSem;
}



hipExternalSemaphore_t importVulkanSemaphoreObjectFromNTHandle(HANDLE handle) {
    hipExternalSemaphore_t extSem = NULL;
    hipExternalSemaphoreHandleDesc desc = {};
    memset(&desc, 0, sizeof(desc));
    desc.type = hipExternalSemaphoreHandleTypeOpaqueWin32;
    desc.handle.win32.handle = handle;
    hipImportExternalSemaphore(&extSem, &desc);
    // Input parameter 'handle' should be closed if it's not needed anymore
    CloseHandle(handle);
    return extSem;
}

hipExternalSemaphore_t importVulkanSemaphoreObjectFromNamedNTHandle(LPCWSTR name) {
    hipExternalSemaphore_t extSem = NULL;
    hipExternalSemaphoreHandleDesc desc = {};
    memset(&desc, 0, sizeof(desc));
    desc.type = hipExternalSemaphoreHandleTypeOpaqueWin32;
    desc.handle.win32.name = (void *)name;
    hipImportExternalSemaphore(&extSem, &desc);
    return extSem;
}


hipExternalSemaphore_t importVulkanSemaphoreObjectFromKMTHandle(HANDLE handle)
{
    hipExternalSemaphore_t extSem = NULL;
    hipExternalSemaphoreHandleDesc desc = {};
    memset(&desc, 0, sizeof(desc));
    desc.type = hipExternalSemaphoreHandleTypeOpaqueWin32Kmt;
    desc.handle.win32.handle = (void *)handle;
    hipImportExternalSemaphore(&extSem, &desc);
    return extSem;
}

void signalExternalSemaphore(hipExternalSemaphore_t extSem, hipStream_t stream) {
    hipExternalSemaphoreSignalParams params = {};
    memset(&params, 0, sizeof(params));
    hipSignalExternalSemaphoresAsync(&extSem, &params, 1, stream);
}

void waitExternalSemaphore(hipExternalSemaphore_t extSem, hipStream_t stream)
{
    hipExternalSemaphoreWaitParams params = {};
    memset(&params, 0, sizeof(params));
    hipWaitExternalSemaphoresAsync(&extSem, &params, 1, stream);
}
