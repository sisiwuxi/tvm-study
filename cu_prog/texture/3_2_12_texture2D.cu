#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cu_prog.h"
// struct hipTextureDesc
// {
//     enum hipTextureAddressMode addressMode[3];
//     enum hipTextureFilterMode  filterMode;
//     enum hipTextureReadMode    readMode;
//     int                         sRGB;
//     int                         normalizedCoords;
//     unsigned int                maxAnisotropy;
//     enum hipTextureFilterMode  mipmapFilterMode;
//     float                       mipmapLevelBias;
//     float                       minMipmapLevelClamp;
//     float                       maxMipmapLevelClamp;
// };
// 2D float texture
texture<float, hipTextureType2D, hipReadModeElementType> texRef;

// Simple transformation kernel
__global__ void transformKernel(float* output,
                                int width, int height,
                                float theta) 
{
    // Calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = x / (float)width;
    float v = y / (float)height;

    // Transform coordinates
    u -= 0.5f;
    v -= 0.5f;
    float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
    float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;


    // Read from texture and write to global memory
    output[y * width + x] = tex2D(texRef, tu, tv);
}

// Host code
int main()
{
    const int height = 32;//1024;
    const int width = 32;//1024;
    float angle = 0.5;
    // Allocate and set some host data
    float *h_data = (float *)std::malloc(sizeof(float) * width * height);
    for (int i = 0; i < height * width; ++i)
        h_data[i] = i;
    printMatrix(h_data, width, height);
    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc =
               hipCreateChannelDesc(32, 0, 0, 0,
                                     hipChannelFormatKindFloat);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);

    // Copy to device memory some data located at address h_data
    // in host memory 
    size_t size = width * height * sizeof(float);
    hipMemcpyToArray(cuArray, 0, 0, h_data, size,
                      hipMemcpyHostToDevice);

    // Set texture reference parameters
    texRef.addressMode[0] = hipAddressModeWrap;
    texRef.addressMode[1] = hipAddressModeWrap;
    texRef.filterMode     = hipFilterModeLinear;
    texRef.normalized     = true;

    // Bind the array to the texture reference
    hipBindTextureToArray(texRef, cuArray, channelDesc);

    // Allocate result of transformation in device memory
    float* output;
    hipMalloc(&output, size);

    // Invoke kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width  + dimBlock.x - 1) / dimBlock.x,
                 (height + dimBlock.y - 1) / dimBlock.y);
    transformKernel<<<dimGrid, dimBlock>>>(output, width, height,
                                           angle);
    // Copy data from device back to host
    // hipMemcpyFromArray(h_data, cuArray, 0, 0, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_data, output, size, hipMemcpyDeviceToHost);
    // Free device memory
    hipFreeArray(cuArray);
    hipFree(output);
    printMatrix(h_data, width, height);
    // Free host memory
    free(h_data);
    return 0;
}