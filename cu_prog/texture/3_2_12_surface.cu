#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cu_prog.h"
// Simple copy kernel
__global__ void copyKernel(hipSurfaceObject_t inputSurfObj,
                           hipSurfaceObject_t outputSurfObj,
                           int width, int height) 
{
    // Calculate surface coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        uchar4 data;
        // Read from input surface
        surf2Dread(&data,  inputSurfObj, x * 4, y);
        // Write to output surface
        surf2Dwrite(data, outputSurfObj, x * 4, y);
    }
}

void low_level_api() {
    const surface<void, cudaSurfaceType2D> surfRef;
    const surfaceReference* surfRefPtr;
    // cudaGetSurfaceReference(&surfRefPtr, "surfRef");
    cudaGetSurfaceReference(&surfRefPtr, (const void *)&surfRef);
    hipChannelFormatDesc channelDesc;
    hipArray_const_t cuArray;
    hipGetChannelDesc(&channelDesc, cuArray);
    // cudaBindSurfaceToArray(surfRef, cuArray, &channelDesc);
    cudaBindSurfaceToArray(surfRef, cuArray);
}

void high_level_api() {
    surface<void, cudaSurfaceType2D> surfRef;
    hipArray_const_t cuArray;
    cudaBindSurfaceToArray(surfRef, cuArray);
}

// Host code
int main()
{
    const int height = 32;//1024;
    const int width = 32;//1024;

    // Allocate and set some host data
    unsigned char *h_data =
        (unsigned char *)std::malloc(sizeof(unsigned char) * width * height * 4);
    for (int i = 0; i < height * width * 4; ++i)
        h_data[i] = i;
    printSurface(h_data, width, height, 4);
    // Allocate CUDA arrays in device memory
    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    hipArray_t cuInputArray;
    hipMallocArray(&cuInputArray, &channelDesc, width, height,
                    hipArraySurfaceLoadStore);
    hipArray_t cuOutputArray;
    hipMallocArray(&cuOutputArray, &channelDesc, width, height,
                    hipArraySurfaceLoadStore);

    // Set pitch of the source (the width in memory in bytes of the 2D array
    // pointed to by src, including padding), we dont have any padding
    const size_t spitch = 4 * width * sizeof(unsigned char);
    // Copy data located at address h_data in host memory to device memory
    hipMemcpy2DToArray(cuInputArray, 0, 0, h_data, spitch,
                        4 * width * sizeof(unsigned char), height,
                        hipMemcpyHostToDevice);

    // Specify surface
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    // Create the surface objects
    resDesc.res.array.array = cuInputArray;
    hipSurfaceObject_t inputSurfObj = 0;
    hipCreateSurfaceObject(&inputSurfObj, &resDesc);
    resDesc.res.array.array = cuOutputArray;
    hipSurfaceObject_t outputSurfObj = 0;
    hipCreateSurfaceObject(&outputSurfObj, &resDesc);

    // Invoke kernel
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x,
                    (height + threadsperBlock.y - 1) / threadsperBlock.y);
    copyKernel<<<numBlocks, threadsperBlock>>>(inputSurfObj, outputSurfObj, width,height);

    // Copy data from device back to host
    hipMemcpy2DFromArray(h_data, spitch, cuOutputArray, 0, 0,4 * width * sizeof(unsigned char), height,hipMemcpyDeviceToHost);

    // Destroy surface objects
    hipDestroySurfaceObject(inputSurfObj);
    hipDestroySurfaceObject(outputSurfObj);

    // Free device memory
    hipFreeArray(cuInputArray);
    hipFreeArray(cuOutputArray);
    printSurface(h_data, width, height, 4);
    // Free host memory
    free(h_data);

  return 0;
}
