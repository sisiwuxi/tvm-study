#include "hip/hip_runtime.h"
#include<GL/gl.h>
#include<GL/glext.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

IDirect3D9* D3D;
IDirect3DDevice9* device;
struct CUSTOMVERTEX {
    FLOAT x, y, z;
    DWORD color;
};
IDirect3DVertexBuffer9* positionsVB;
struct hipGraphicsResource* positionsVB_CUDA;

__global__ void createVertices(float4* positions, float time, unsigned int width, unsigned int height)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    // Calculate uv coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;
    // Calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u * freq + time)
    * cosf(v * freq + time) * 0.5f;
    // Write positions
    positions[y * width + x] =
    make_float4(u, w, v, __int_as_float(0xff00ff00));
}

void Render()
{
    // Map vertex buffer for writing from CUDA
    float4* positions;
    hipGraphicsMapResources(1, &positionsVB_CUDA, 0);
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void**)&positions,&num_bytes,positionsVB_CUDA));
    // Execute kernel
    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);
    createVertices<<<dimGrid, dimBlock>>>(positions, time,width, height);
    // Unmap vertex buffer
    hipGraphicsUnmapResources(1, &positionsVB_CUDA, 0);
    // Draw and present
    ...
}

void releaseVB()
{
    hipGraphicsUnregisterResource(positionsVB_CUDA);
    positionsVB->Release();
}


int main()
{
    int dev;
    // Initialize Direct3D
    D3D = Direct3DCreate9Ex(D3D_SDK_VERSION);
    // Get a CUDA-enabled adapter
    unsigned int adapter = 0;
    for (; adapter < g_pD3D->GetAdapterCount(); adapter++) {
        D3DADAPTER_IDENTIFIER9 adapterId;
        g_pD3D->GetAdapterIdentifier(adapter, 0, &adapterId);
        if (cudaD3D9GetDevice(&dev, adapterId.DeviceName) == hipSuccess)
            break;
    }
    // Create device
    ...
    D3D->CreateDeviceEx(adapter, D3DDEVTYPE_HAL, hWnd,D3DCREATE_HARDWARE_VERTEXPROCESSING,&params, NULL, &device);
    // Use the same device
    hipSetDevice(dev);
    // Create vertex buffer and register it with CUDA
    unsigned int size = width * height * sizeof(CUSTOMVERTEX);
    device->CreateVertexBuffer(size, 0, D3DFVF_CUSTOMVERTEX,D3DPOOL_DEFAULT, &positionsVB, 0);
    cudaGraphicsD3D9RegisterResource(&positionsVB_CUDA,positionsVB,hipGraphicsRegisterFlagsNone);
    cudaGraphicsResourceSetMapFlags(positionsVB_CUDA,cudaGraphicsMapFlagsWriteDiscard);
    // Launch rendering loop
    while (...) {
        ...
        Render();
        ...
    }
    ...
}