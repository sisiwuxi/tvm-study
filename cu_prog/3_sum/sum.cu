
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "sys/time.h"

__global__
void sum_global(float *input, float *output, int thread)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int idx = bid*blockDim.x + tid;

    for (int i=thread/2; i>0; i>>=1)
    {
        if (idx < i)
        {
            input[idx] += input[idx + i];
        }
        __syncthreads();
    }
    if (idx == 0)
    {
        output[0] = input[0];
    }
}

__global__
void sum_shared(float *a, float *b, int thread)
{
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int idx = bid*blockDim.x + tid;
    
    extern __shared__ float sData[];
    sData[idx] = a[idx];
    __syncthreads();
    for (int i=thread/2; i>0; i>>=1)
    {
        if (idx < i)
        {
            sData[idx] += sData[idx + i];
        }
        __syncthreads();
    }
    if (idx == 0)
    {
        b[0] = sData[0];
    }
}

void cpuSum(float *a, float *b, int thread)
{
    b[0] = 0;
    for (int i=0; i<thread; ++i)
    {
        b[0] += a[i];
    }
}

int main()
{
    int maxThreadsPerBlock = 1024;
    int thread = 8192;//102400;//8192;//8194;//4096;//1024;
    float a[thread];
    for (int i=0; i<thread; ++i)
    {
        // a[i] = i*(i+1);
        a[i] = 1.0f;
    }
    float *aGpu;
    hipMalloc((void**)&aGpu, thread*sizeof(float));
    hipMemcpy(aGpu, a, thread*sizeof(float), hipMemcpyHostToDevice);

    float b[1];
    int iterations = 10000;
    struct timeval startTime, endTime;


    gettimeofday(&startTime, NULL);
    for (int i=0; i<thread; ++i)
    {
        cpuSum(a, b, thread);
    }
    gettimeofday(&endTime, NULL);
    long int latency_cpu = (endTime.tv_sec - startTime.tv_sec)*1000000 + (endTime.tv_usec - startTime.tv_usec);    
    printf("cpuSum:%f latency_cpu = %ld\n", b[0], latency_cpu);

    int Dg = (thread + maxThreadsPerBlock - 1)/maxThreadsPerBlock;

    float *bGpu_global;
    hipMalloc((void**)&bGpu_global, 1*sizeof(float));
    gettimeofday(&startTime, NULL);
    for (int i=0; i<iterations; ++i)
    {
        hipMemcpy(aGpu, a, thread*sizeof(float), hipMemcpyHostToDevice);
        sum_global<<<Dg, maxThreadsPerBlock>>>(aGpu, bGpu_global, thread);
    }
    gettimeofday(&endTime, NULL);
    long int latency_global = (endTime.tv_sec - startTime.tv_sec)*1000000 + (endTime.tv_usec - startTime.tv_usec);    
    hipMemcpy(b, bGpu_global, 1*sizeof(float), hipMemcpyDeviceToHost);
    printf("sum_global:%f latency_global = %ld\n", b[0], latency_global);
    

    hipMemcpy(aGpu, a, thread*sizeof(float), hipMemcpyHostToDevice);


    float *bGpu_shared;
    hipMalloc((void**)&bGpu_shared, 1*sizeof(float));
    gettimeofday(&startTime, NULL);
    for (int i=0; i<iterations; ++i)
    {
        sum_shared<<<Dg, maxThreadsPerBlock, maxThreadsPerBlock*sizeof(float)>>>(aGpu, bGpu_shared, thread);
    }
    gettimeofday(&endTime, NULL);
    long int latency_shared = (endTime.tv_sec - startTime.tv_sec)*1000000 + (endTime.tv_usec - startTime.tv_usec);
    hipMemcpy(b, bGpu_shared, 1*sizeof(float), hipMemcpyDeviceToHost);
    printf("sum_shared:%f latency_shared = %ld\n", b[0], latency_shared);


    return 0;
}