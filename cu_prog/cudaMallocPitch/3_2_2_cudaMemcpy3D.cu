#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "cu_prog.h"
#include "gputimer.h"

__constant__ float constData[256];

__device__ float devData;

__device__ float* devPointer;

// Device code for hipMemcpy3D
__global__ void MyKernel(hipPitchedPtr devPitchedPtr,
                         int width, int height, int depth)
{
    char* devPtr = (char*)devPitchedPtr.ptr;
    size_t pitch = devPitchedPtr.pitch;
    size_t slicePitch = pitch * height;
    for (int z = 0; z < depth; ++z) {
        char* slice = devPtr + z * slicePitch;
        for (int y = 0; y < height; ++y) {
            float* row = (float*)(slice + y * pitch);
            for (int x = 0; x < width; ++x) {
                float element = row[x];
            }
        }
    }
}

int main()
{
    // Host code
    int width = 64, height = 64, depth = 64;
    hipExtent extent = make_hipExtent(width * sizeof(float), height, depth);
    hipPitchedPtr devPitchedPtr;
    hipMalloc3D(&devPitchedPtr, extent);
    MyKernel<<<100, 512>>>(devPitchedPtr, width, height, depth);

    float data[256];
    hipMemcpyToSymbol(HIP_SYMBOL(constData), data, sizeof(data));
    hipMemcpyFromSymbol(data, HIP_SYMBOL(constData), sizeof(data));
    float value = 3.14f;
    hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));
    float* ptr;
    hipMalloc(&ptr, 256 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(devPointer), &ptr, sizeof(ptr));

    free(ptr);
	// CHECK(hipFree(devPitchedPtr.ptr)); coredump?
}



