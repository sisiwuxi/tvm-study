
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void MyKernel(float *c, float *a, const int size)
{
    return;
}

// CPU callback
void CUDART_CB MyCallback(hipStream_t stream, hipError_t status, void *data){
    printf("Inside callback %d\n", (size_t)data);
    return;
}

void create_graph() {
    /*
            A
          /   \
        B       C
          \   /
            D
    */

    // // Create the graph - it starts out empty
    // cudaGraphCreate(&graph, 0);

    // // For the purpose of this example, we'll create
    // // the nodes separately from the dependencies to
    // // demonstrate that it can be done in two stages.
    // // Note that dependencies can also be specified 
    // // at node creation. 
    // cudaGraphAddKernelNode(&a, graph, NULL, 0, &nodeParams);
    // cudaGraphAddKernelNode(&b, graph, NULL, 0, &nodeParams);
    // cudaGraphAddKernelNode(&c, graph, NULL, 0, &nodeParams);
    // cudaGraphAddKernelNode(&d, graph, NULL, 0, &nodeParams);

    // // Now set up dependencies on each node
    // cudaGraphAddDependencies(graph, &a, &b, 1);     // A->B
    // cudaGraphAddDependencies(graph, &a, &c, 1);     // A->C
    // cudaGraphAddDependencies(graph, &b, &d, 1);     // B->D
    // cudaGraphAddDependencies(graph, &c, &d, 1);     // C->D

    // cudaGraph_t graph;
    // cudaStreamBeginCapture(stream);
    // kernel_A<<< ..., stream >>>(...);
    // kernel_B<<< ..., stream >>>(...);
    // libraryCall(stream);
    // kernel_C<<< ..., stream >>>(...);

    // cudaStreamEndCapture(stream, &graph);
    // cudaStreamIsCapturing()

    // // stream1 is the origin stream
    // cudaStreamBeginCapture(stream1);
    // kernel_A<<< ..., stream1 >>>(...);
    // // Fork into stream2
    // cudaEventRecord(event1, stream1);
    // cudaStreamWaitEvent(stream2, event1);
    // kernel_B<<< ..., stream1 >>>(...);
    // kernel_C<<< ..., stream2 >>>(...);
    // // Join stream2 back to origin stream (stream1)
    // cudaEventRecord(event2, stream2);
    // cudaStreamWaitEvent(stream1, event2);
    // kernel_D<<< ..., stream1 >>>(...);
    // // End capture in the origin stream
    // cudaStreamEndCapture(stream1, &graph);
    // // stream1 and stream2 no longer in capture mode 
    return;
}

// void update_graph() {
//     cudaGraphExec_t graphExec = NULL;

//     for (int i = 0; i < 10; i++) {
//         cudaGraph_t graph;
//         cudaGraphExecUpdateResult updateResult;
//         cudaGraphNode_t errorNode;

//         // In this example we use stream capture to create the graph.
//         // You can also use the Graph API to produce a graph.
//         cudaStreamBeginCapture(stream, cudaStreamCaptureModeGlobal);

//         // Call a user-defined, stream based workload, for example
//         do_cuda_work(stream);

//         cudaStreamEndCapture(stream, &graph);

//         // If we've already instantiated the graph, try to update it directly
//         // and avoid the instantiation overhead
//         if (graphExec != NULL) {
//             // If the graph fails to update, errorNode will be set to the
//             // node causing the failure and updateResult will be set to a
//             // reason code.
//             cudaGraphExecUpdate(graphExec, graph, &errorNode, &updateResult);
//         }

//         // Instantiate during the first iteration or whenever the update
//         // fails for any reason
//         if (graphExec == NULL || updateResult != cudaGraphExecUpdateSuccess) {

//             // If a previous update failed, destroy the cudaGraphExec_t
//             // before re-instantiating it
//             if (graphExec != NULL) {
//                 cudaGraphExecDestroy(graphExec);
//             }   
//             // Instantiate graphExec from graph. The error node and
//             // error message parameters are unused here.
//             cudaGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
//         }   

//         cudaGraphDestroy(graph);
//         cudaGraphLaunch(graphExec, stream);
//         cudaStreamSynchronize(stream);
//     }
// }

void update_node_update() {
    // cudaGraphExecKernelNodeSetParams()
    // cudaGraphExecMemcpyNodeSetParams()
    // cudaGraphExecMemsetNodeSetParams()
    // cudaGraphExecHostNodeSetParams()
    // cudaGraphExecChildGraphNodeSetParams()
    // cudaGraphExecEventRecordNodeSetEvent()
    // cudaGraphExecEventWaitNodeSetEvent()
    // cudaGraphExecExternalSemaphoresSignalNodeSetParams()
    // cudaGraphExecExternalSemaphoresWaitNodeSetParams()

}

void event() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    float* inputDev, *outputDev;
    float* inputHost, *outputHost;
    int size = 16;
    hipStream_t stream[2];
    for (int i = 0; i < 2; ++i) {
        hipMemcpyAsync(inputDev + i * size, inputHost + i * size,
                        size, hipMemcpyHostToDevice, stream[i]);
        MyKernel<<<100, 512, 0, stream[i]>>>
                (outputDev + i * size, inputDev + i * size, size);
        hipMemcpyAsync(outputHost + i * size, outputDev + i * size,
                        size, hipMemcpyDeviceToHost, stream[i]);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void multiDevice() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device %d has compute capability %d.%d.\n",
            device, deviceProp.major, deviceProp.minor);
    }

    size_t size = 1024 * sizeof(float);
    hipSetDevice(0);            // Set device 0 as current
    float* p0;
    hipMalloc(&p0, size);       // Allocate memory on device 0
    MyKernel<<<1000, 128>>>(p0,p0,size); // Launch kernel on device 0
    hipSetDevice(1);            // Set device 1 as current
    float* p1;
    hipMalloc(&p1, size);       // Allocate memory on device 1
    MyKernel<<<1000, 128>>>(p1,p1,size); // Launch kernel on device 1


    hipSetDevice(0);               // Set device 0 as current
    hipStream_t s0;
    hipStreamCreate(&s0);          // Create stream s0 on device 0
    MyKernel<<<100, 64, 0, s0>>>(p0,p0,size); // Launch kernel on device 0 in s0
    hipSetDevice(1);               // Set device 1 as current
    hipStream_t s1;
    hipStreamCreate(&s1);          // Create stream s1 on device 1
    MyKernel<<<100, 64, 0, s1>>>(p0,p0,size); // Launch kernel on device 1 in s1

    // This kernel launch will fail:
    MyKernel<<<100, 64, 0, s0>>>(p0,p0,size); // Launch kernel on device 1 in s0

    // cudaEventRecord
    // cudaEventElapsedTime
    // cudaEventSynchronize
    // cudaEventQuery
    // cudaStreamWaitEvent

    hipSetDevice(0);                   // Set device 0 as current
    // float* p0;
    // size_t size = 1024 * sizeof(float);
    hipMalloc(&p0, size);              // Allocate memory on device 0
    MyKernel<<<1000, 128>>>(p0,p0,size);        // Launch kernel on device 0
    hipSetDevice(1);                   // Set device 1 as current
    hipDeviceEnablePeerAccess(0, 0);   // Enable peer-to-peer access
                                        // with device 0

    // Launch kernel on device 1
    // This kernel launch can access memory on device 0 at address p0
    MyKernel<<<1000, 128>>>(p0,p0,size);


    hipSetDevice(0);                   // Set device 0 as current
    // float* p0;
    // size_t size = 1024 * sizeof(float);
    hipMalloc(&p0, size);              // Allocate memory on device 0
    hipSetDevice(1);                   // Set device 1 as current
    // float* p1;
    hipMalloc(&p1, size);              // Allocate memory on device 1
    hipSetDevice(0);                   // Set device 0 as current
    MyKernel<<<1000, 128>>>(p0,p0,size);        // Launch kernel on device 0
    hipSetDevice(1);                   // Set device 1 as current
    hipMemcpyPeer(p1, 1, p0, 0, size); // Copy p0 to p1
    MyKernel<<<1000, 128>>>(p0,p0,size);        // Launch kernel on device 1


}







int main()
{
    hipStream_t stream[2];
    for (int i = 0; i < 2; ++i) {
        hipStreamCreate(&stream[i]);
    }
        
    float* hostPtr;
    int size = 16;
    hipHostMalloc(&hostPtr, 2 * size, hipHostMallocDefault);
    float* inputDevPtr;
    float* outputDevPtr;

    for (int i = 0; i < 2; ++i) {
        hipMemcpyAsync(inputDevPtr + i * size, hostPtr + i * size, size, hipMemcpyHostToDevice, stream[i]);
        MyKernel <<<100, 512, 0, stream[i]>>>
            (outputDevPtr + i * size, inputDevPtr + i * size, size);
        hipMemcpyAsync(hostPtr + i * size, outputDevPtr + i * size,
                        size, hipMemcpyDeviceToHost, stream[i]);
    }
    // overlap
    for (int i = 0; i < 2; ++i) {
        hipMemcpyAsync(inputDevPtr + i * size, hostPtr + i * size, size, hipMemcpyHostToDevice, stream[i]);
    }
    for (int i = 0; i < 2; ++i)
    {
        MyKernel<<<100, 512, 0, stream[i]>>>
            (outputDevPtr + i * size, inputDevPtr + i * size, size);
        for (int i = 0; i < 2; ++i)
        hipMemcpyAsync(hostPtr + i * size, outputDevPtr + i * size,
                        size, hipMemcpyDeviceToHost, stream[i]);
    }

    float* devPtrIn[2];
    float* hostPtrIn[2];
    float* devPtrOut[2];
    for (size_t i = 0; i < 2; ++i) {
        hipMemcpyAsync(devPtrIn[i], hostPtrIn[i], size, hipMemcpyHostToDevice, stream[i]);
        MyKernel<<<100, 512, 0, stream[i]>>>(devPtrOut[i], devPtrIn[i], size);
        hipMemcpyAsync(hostPtrIn[i], devPtrOut[i], size, hipMemcpyDeviceToHost, stream[i]);
        // cudaLaunchHostFunc(stream[i], MyCallback, (void*)i);
    }

    // get the range of stream priorities for this device
    int priority_high, priority_low;
    hipDeviceGetStreamPriorityRange(&priority_low, &priority_high);
    // create streams with highest and lowest available priorities
    hipStream_t st_high, st_low;
    hipStreamCreateWithPriority(&st_high, hipStreamNonBlocking, priority_high);
    hipStreamCreateWithPriority(&st_low, hipStreamNonBlocking, priority_low);

    for (int i = 0; i < 2; ++i)
    {
        hipStreamDestroy(stream[i]);
    }
    hipDeviceSynchronize();
    hipEvent_t event = 0;
    unsigned int flag = 0;
    hipStreamWaitEvent(st_high, event, flag);
    hipStreamQuery(st_high);
}

