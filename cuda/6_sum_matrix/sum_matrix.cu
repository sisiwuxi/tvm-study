#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"
void sumMatrix2D_CPU(float * MatA,float * MatB,float * MatC,int nx,int ny)
{
  float * a=MatA;
  float * b=MatB;
  float * c=MatC;
  for(int j=0;j<ny;j++)
  {
    for(int i=0;i<nx;i++)
    {
      c[i]=a[i]+b[i];
    }
    c+=nx;
    b+=nx;
    a+=nx;
  }
}
__global__ void sumMatrix(float * MatA,float * MatB,float * MatC,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx=ix+iy*ny;
    if (ix<nx && iy<ny)
    {
      MatC[idx]=MatA[idx]+MatB[idx];
    }
}

int main(int argc,char** argv)
{
  printf("strating...\n");
  initDevice(0);
  int nx=1<<12;
  int ny=1<<12;
  int nxy=nx*ny;
  int nBytes=nxy*sizeof(float);

  //Malloc
  float* A_host=(float*)malloc(nBytes);
  float* B_host=(float*)malloc(nBytes);
  float* C_host=(float*)malloc(nBytes);
  float* C_from_gpu=(float*)malloc(nBytes);
  initialData(A_host,nxy);
  initialData(B_host,nxy);

  //hipMalloc
  float *A_dev=NULL;
  float *B_dev=NULL;
  float *C_dev=NULL;
  CHECK(hipMalloc((void**)&A_dev,nBytes));
  CHECK(hipMalloc((void**)&B_dev,nBytes));
  CHECK(hipMalloc((void**)&C_dev,nBytes));


  CHECK(hipMemcpy(A_dev,A_host,nBytes,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(B_dev,B_host,nBytes,hipMemcpyHostToDevice));


  // cpu compute
  hipMemcpy(C_from_gpu,C_dev,nBytes,hipMemcpyDeviceToHost);
  double iStart=cpuSecond();
  sumMatrix2D_CPU(A_host,B_host,C_host,nx,ny);
  double iElaps=cpuSecond()-iStart;
  printf("CPU Execution Time elapsed %f sec\n",iElaps);

  // 2d block and 2d grid
  int dimx=32;
  int dimy=32;
  dim3 block_0(dimx,dimy);
  dim3 grid_0((nx-1)/block_0.x+1,(ny-1)/block_0.y+1);
  iStart=cpuSecond();
  sumMatrix<<<grid_0,block_0>>>(A_dev,B_dev,C_dev,nx,ny);
  CHECK(hipDeviceSynchronize());
  iElaps=cpuSecond()-iStart;
  printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f sec\n",
        grid_0.x,grid_0.y,block_0.x,block_0.y,iElaps);
  CHECK(hipMemcpy(C_from_gpu,C_dev,nBytes,hipMemcpyDeviceToHost));
  checkResult(C_host,C_from_gpu,nxy);

  // 1d block and 1d grid
  dimx=32;
  dim3 block_1(dimx);
  dim3 grid_1((nxy-1)/block_1.x+1);
  iStart=cpuSecond();
  sumMatrix<<<grid_1,block_1>>>(A_dev,B_dev,C_dev,nx*ny ,1);
  CHECK(hipDeviceSynchronize());
  iElaps=cpuSecond()-iStart;
  printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f sec\n",
        grid_1.x,grid_1.y,block_1.x,block_1.y,iElaps);
  CHECK(hipMemcpy(C_from_gpu,C_dev,nBytes,hipMemcpyDeviceToHost));
  checkResult(C_host,C_from_gpu,nxy);

  // 2d block and 1d grid
  dimx=32;
  dim3 block_2(dimx);
  dim3 grid_2((nx-1)/block_2.x+1,ny);
  iStart=cpuSecond();
  sumMatrix<<<grid_2,block_2>>>(A_dev,B_dev,C_dev,nx,ny);
  CHECK(hipDeviceSynchronize());
  iElaps=cpuSecond()-iStart;
  printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f sec\n",
        grid_2.x,grid_2.y,block_2.x,block_2.y,iElaps);
  CHECK(hipMemcpy(C_from_gpu,C_dev,nBytes,hipMemcpyDeviceToHost));
  checkResult(C_host,C_from_gpu,nxy);


  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);
  free(A_host);
  free(B_host);
  free(C_host);
  free(C_from_gpu);
  hipDeviceReset();
  return 0;
}
