#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"

struct naiveStruct{
    float a;
    float b;
};
void sumArrays(float * a,float * b,float * res,const int size)
{

    for(int i=0;i<size;i++)
    {
        res[i]=a[i]+b[i];
    }

}
__global__ void sumArraysGPU(float*a,float*b,struct naiveStruct* res,int n)
{
  //int i=threadIdx.x;
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i<n)
    res[i].a=a[i]+b[i];
}
void checkResult_struct(float* res_h,struct naiveStruct*res_from_gpu_h,int nElem)
{
    for(int i=0;i<nElem;i++)
        if (res_h[i]!=res_from_gpu_h[i].a)
        {
            printf("check fail!\n");
            exit(0);
        }
    printf("result check success!\n");
}
int main(int argc,char **argv)
{
  int dev = 0;
  hipSetDevice(dev);

  int nElem=1<<18;
  int offset=0;
  if(argc>=2)
    offset=atoi(argv[1]);
  printf("Vector size:%d\n",nElem);
  int nByte=sizeof(float)*nElem;
  int nByte_struct=sizeof(struct naiveStruct)*nElem;
  float *a_h=(float*)malloc(nByte);
  float *b_h=(float*)malloc(nByte);
  float *res_h=(float*)malloc(nByte_struct);
  struct naiveStruct *res_from_gpu_h=(struct naiveStruct*)malloc(nByte_struct);
  memset(res_h,0,nByte);
  memset(res_from_gpu_h,0,nByte);

  float *a_d,*b_d;
  struct naiveStruct* res_d;
  CHECK(hipMalloc((float**)&a_d,nByte));
  CHECK(hipMalloc((float**)&b_d,nByte));
  CHECK(hipMalloc((struct naiveStruct**)&res_d,nByte_struct));
  CHECK(hipMemset(res_d,0,nByte_struct));
  initialData(a_h,nElem);
  initialData(b_h,nElem);

  CHECK(hipMemcpy(a_d,a_h,nByte,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(b_d,b_h,nByte,hipMemcpyHostToDevice));

  dim3 block(1024);
  dim3 grid(nElem/block.x);
  double iStart,iElaps;
  iStart=cpuSecond();
  sumArraysGPU<<<grid,block>>>(a_d,b_d,res_d,nElem);
  hipDeviceSynchronize();
  iElaps=cpuSecond()-iStart;
  CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte_struct,hipMemcpyDeviceToHost));
  printf("Execution configuration<<<%d,%d>>> Time elapsed %f sec\n",grid.x,block.x,iElaps);


  sumArrays(a_h,b_h,res_h,nElem);

  checkResult_struct(res_h,res_from_gpu_h,nElem);
  hipFree(a_d);
  hipFree(b_d);
  hipFree(res_d);

  free(a_h);
  free(b_h);
  free(res_h);
  free(res_from_gpu_h);

  return 0;
}
