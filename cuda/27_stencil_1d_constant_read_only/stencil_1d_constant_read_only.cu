#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"
#define TEMPLATE_SIZE 9
#define TEMP_RADIO_SIZE (TEMPLATE_SIZE/2)
#define BDIM 32

__constant__ float coef[TEMP_RADIO_SIZE];//if in midle of the program will be error
void convolution(float *in,float *out,float* template_,const unsigned int array_size)
{
    for(int i=TEMP_RADIO_SIZE;i<array_size-TEMP_RADIO_SIZE;i++)
    {
        for(int j=1;j<=TEMP_RADIO_SIZE;j++)
        {
            out[i]+=template_[j-1]*(in[i+j]-in[i-j]);
        }

        //printf("%d:CPU :%lf\n",i,out[i]);
    }

}

__global__ void stencil_1d(float * in,float * out)
{
    __shared__ float smem[BDIM+2*TEMP_RADIO_SIZE];
    int idx=threadIdx.x+blockDim.x*blockIdx.x;
    int sidx=threadIdx.x+TEMP_RADIO_SIZE;
    smem[sidx]=in[idx];

    if (threadIdx.x<TEMP_RADIO_SIZE)

    {
        if(idx>TEMP_RADIO_SIZE)
            smem[sidx-TEMP_RADIO_SIZE]=in[idx-TEMP_RADIO_SIZE];
        if(idx<gridDim.x*blockDim.x-BDIM)
            smem[sidx+BDIM]=in[idx+BDIM];

    }

    __syncthreads();
    if (idx<TEMP_RADIO_SIZE||idx>=gridDim.x*blockDim.x-TEMP_RADIO_SIZE)
        return;
    float temp=.0f;
    #pragma unroll
    for(int i=1;i<=TEMP_RADIO_SIZE;i++)
    {
        temp+=coef[i-1]*(smem[sidx+i]-smem[sidx-i]);
    }
    out[idx]=temp;
    //printf("%d:GPU :%lf,\n",idx,temp);
}
//read only
__global__ void stencil_1d_readonly(float * in,float * out,const float* __restrict__ dcoef)
{
    __shared__ float smem[BDIM+2*TEMP_RADIO_SIZE];
    int idx=threadIdx.x+blockDim.x*blockIdx.x;
    int sidx=threadIdx.x+TEMP_RADIO_SIZE;
    smem[sidx]=in[idx];

    if (threadIdx.x<TEMP_RADIO_SIZE)

    {
        if(idx>TEMP_RADIO_SIZE)
            smem[sidx-TEMP_RADIO_SIZE]=in[idx-TEMP_RADIO_SIZE];
        if(idx<gridDim.x*blockDim.x-BDIM)
            smem[sidx+BDIM]=in[idx+BDIM];

    }

    __syncthreads();
    if (idx<TEMP_RADIO_SIZE||idx>=gridDim.x*blockDim.x-TEMP_RADIO_SIZE)
        return;
    float temp=.0f;
    #pragma unroll
    for(int i=1;i<=TEMP_RADIO_SIZE;i++)
    {
        temp+=dcoef[i-1]*(smem[sidx+i]-smem[sidx-i]);
    }
    out[idx]=temp;
    //printf("%d:GPU :%lf,\n",idx,temp);
}

int main(int argc,char** argv)
{
    printf("strating...\n");
    initDevice(0);
    int dimx=BDIM;
    unsigned int nxy=1<<16;
    int nBytes=nxy*sizeof(float);


    //Malloc
    float* in_host=(float*)malloc(nBytes);
    float* out_gpu=(float*)malloc(nBytes);
    float* out_cpu=(float*)malloc(nBytes);
    memset(out_cpu,0,nBytes);
    initialData(in_host,nxy);

    //hipMalloc
    float *in_dev=NULL;
    float *out_dev=NULL;

    initialData(in_host,nxy);
    float templ_[]={-1.0,-2.0,2.0,1.0};
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(coef),templ_,TEMP_RADIO_SIZE*sizeof(float)));

    CHECK(hipMalloc((void**)&in_dev,nBytes));
    CHECK(hipMalloc((void**)&out_dev,nBytes));
    CHECK(hipMemcpy(in_dev,in_host,nBytes,hipMemcpyHostToDevice));
    CHECK(hipMemset(out_dev,0,nBytes));



    // cpu compute
    double iStart=cpuSecond();
    convolution(in_host,out_cpu,templ_,nxy);
    double iElaps=cpuSecond()-iStart;
    //printf("CPU Execution Time elapsed %f sec\n",iElaps);

    // stencil 1d
    dim3 block(dimx);
    dim3 grid((nxy-1)/block.x+1);
    stencil_1d<<<grid,block>>>(in_dev,out_dev);
    CHECK(hipDeviceSynchronize());
    iElaps=cpuSecond()-iStart;
    printf("stencil_1d Time elapsed %f sec\n",iElaps);
    CHECK(hipMemcpy(out_gpu,out_dev,nBytes,hipMemcpyDeviceToHost));
    checkResult(out_cpu,out_gpu,nxy);
    CHECK(hipMemset(out_dev,0,nBytes));
    // stencil 1d read only
    float * dcoef_ro;
    CHECK(hipMalloc((void**)&dcoef_ro,TEMP_RADIO_SIZE * sizeof(float)));
    CHECK(hipMemcpy(dcoef_ro,templ_,TEMP_RADIO_SIZE * sizeof(float),hipMemcpyHostToDevice));
    stencil_1d_readonly<<<grid,block>>>(in_dev,out_dev,dcoef_ro);
    CHECK(hipDeviceSynchronize());
    iElaps=cpuSecond()-iStart;
    printf("stencil_1d_readonly Time elapsed %f sec\n",iElaps);
    CHECK(hipMemcpy(out_gpu,out_dev,nBytes,hipMemcpyDeviceToHost));
    checkResult(out_cpu,out_gpu,nxy);

    hipFree(dcoef_ro);
    hipFree(in_dev);
    hipFree(out_dev);
    free(out_gpu);
    free(out_cpu);
    free(in_host);
    hipDeviceReset();
    return 0;
}
