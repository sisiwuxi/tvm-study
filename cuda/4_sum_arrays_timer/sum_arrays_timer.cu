#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"



void sumArrays(float * a,float * b,float * res,const int size)
{
  for(int i=0;i<size;i+=4)
  {
    res[i]=a[i]+b[i];
    res[i+1]=a[i+1]+b[i+1];
    res[i+2]=a[i+2]+b[i+2];
    res[i+3]=a[i+3]+b[i+3];
  }
  return;
}
__global__ void sumArraysGPU(float*a,float*b,float*res,int N)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(i < N)
    res[i]=a[i]+b[i];
}
int main(int argc,char **argv)
{
  // set up device
  initDevice(0);

  int nElem=1<<24;
  //int nElem=(1<<24)+1;
  printf("Vector size:%d\n",nElem);
  int nByte=sizeof(float)*nElem;
  float *a_h=(float*)malloc(nByte);
  float *b_h=(float*)malloc(nByte);
  float *res_h=(float*)malloc(nByte);
  float *res_from_gpu_h=(float*)malloc(nByte);
  memset(res_h,0,nByte);
  memset(res_from_gpu_h,0,nByte);

  float *a_d,*b_d,*res_d;
  CHECK(hipMalloc((float**)&a_d,nByte));
  CHECK(hipMalloc((float**)&b_d,nByte));
  CHECK(hipMalloc((float**)&res_d,nByte));

  initialData(a_h,nElem);
  initialData(b_h,nElem);

  CHECK(hipMemcpy(a_d,a_h,nByte,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(b_d,b_h,nByte,hipMemcpyHostToDevice));

  //dim3 block(512);
  dim3 block(nElem);
  dim3 grid((nElem-1)/block.x+1);
  int blocks[] = {1,256,512,1024,2048,4096,8192,16384,32768,65536,131072};
  int blocks_len = sizeof(blocks)/sizeof(blocks[0]);

  for(int i=0;i<blocks_len;i++)
  {
    block.x = blocks[i];
    grid.x = (nElem-1)/block.x+1;
    //timer
    double iStart,iElaps;
    iStart=cpuSecond();
    sumArraysGPU<<<grid,block>>>(a_d,b_d,res_d,nElem);
    CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));
    iElaps=cpuSecond()-iStart;
    printf("Execution configuration<<<%d,%d>>> Time elapsed %f sec\n",grid.x,block.x,iElaps);
    sumArrays(a_h,b_h,res_h,nElem);
    checkResult(res_h,res_from_gpu_h,nElem);
  }
  hipFree(a_d);
  hipFree(b_d);
  hipFree(res_d);

  free(a_h);
  free(b_h);
  free(res_h);
  free(res_from_gpu_h);

  return 0;
}
