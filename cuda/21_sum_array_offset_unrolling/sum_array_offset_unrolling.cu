#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"


void sumArrays(float * a,float * b,float * res,int offset,const int size)
{

    for(int i=0,k=offset;k<size;i++,k++)
    {
        res[i]=a[k]+b[k];
    }

}
__global__ void sumArraysGPU(float*a,float*b,float*res,int offset,int n)
{
  //int i=threadIdx.x;
  int i=blockIdx.x*blockDim.x*4+threadIdx.x;
  int k=i+offset;
  if(k+3*blockDim.x<n)
  {
      res[i]=a[k]+b[k];
      res[i+blockDim.x]=a[k+blockDim.x]+b[k+blockDim.x];
      res[i+blockDim.x*2]=a[k+blockDim.x*2]+b[k+blockDim.x*2];
      res[i+blockDim.x*3]=a[k+blockDim.x*3]+b[k+blockDim.x*3];
  }

}

int main(int argc,char **argv)
{
  int dev = 0;
  hipSetDevice(dev);
  int block_x=512;
  int nElem=1<<18;
  int offset=0;
  if(argc==2)
    offset=atoi(argv[1]);
  else if(argc==3)
    {
        offset=atoi(argv[1]);
        block_x=atoi(argv[2]);
    }
  printf("Vector size:%d\n",nElem);
  int nByte=sizeof(float)*nElem;
  float *a_h=(float*)malloc(nByte);
  float *b_h=(float*)malloc(nByte);
  float *res_h=(float*)malloc(nByte);
  float *res_from_gpu_h=(float*)malloc(nByte);
  memset(res_h,0,nByte);
  memset(res_from_gpu_h,0,nByte);

  float *a_d,*b_d,*res_d;
  CHECK(hipMalloc((float**)&a_d,nByte));
  CHECK(hipMalloc((float**)&b_d,nByte));
  CHECK(hipMalloc((float**)&res_d,nByte));
  CHECK(hipMemset(res_d,0,nByte));
  initialData(a_h,nElem);
  initialData(b_h,nElem);

  CHECK(hipMemcpy(a_d,a_h,nByte,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(b_d,b_h,nByte,hipMemcpyHostToDevice));

  dim3 block(block_x);
  dim3 grid(nElem/block.x);
  double iStart,iElaps;
  iStart=cpuSecond();
  sumArraysGPU<<<grid,block>>>(a_d,b_d,res_d,offset,nElem);
  hipDeviceSynchronize();
  iElaps=cpuSecond()-iStart;

  printf("warmup Time elapsed %f sec\n",iElaps);
  iStart=cpuSecond();
  sumArraysGPU<<<grid,block>>>(a_d,b_d,res_d,offset,nElem);
  hipDeviceSynchronize();
  iElaps=cpuSecond()-iStart;
  CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));
  printf("Execution configuration<<<%d,%d>>> Time elapsed %f sec --offset:%d \n",grid.x,block.x,iElaps,offset);


  sumArrays(a_h,b_h,res_h,offset,nElem);

  checkResult(res_h,res_from_gpu_h,nElem-4*block_x);
  hipFree(a_d);
  hipFree(b_d);
  hipFree(res_d);

  free(a_h);
  free(b_h);
  free(res_h);
  free(res_from_gpu_h);

  return 0;
}
