#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"
void sumMatrix2D_CPU(float * MatA,float * MatB,float * MatC,int nx,int ny)
{
  float * a=MatA;
  float * b=MatB;
  float * c=MatC;
  for(int j=0;j<ny;j++)
  {
    for(int i=0;i<nx;i++)
    {
      c[i]=a[i]+b[i];
    }
    c+=nx;
    b+=nx;
    a+=nx;
  }
}
__global__ void sumMatrix(float * MatA,float * MatB,float * MatC,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx=ix+iy*ny;
    if (ix<nx && iy<ny)
    {
      MatC[idx]=MatA[idx]+MatB[idx];
    }
}

int main(int argc,char** argv)
{
  //printf("strating...\n");
  //initDevice(0);
  int nx=1<<13;
  int ny=1<<13;
  printf("\n nx=%d, ny=%d \n", nx,ny);
  int nxy=nx*ny;
  int nBytes=nxy*sizeof(float);

  //Malloc
  float* A_host=(float*)malloc(nBytes);
  float* B_host=(float*)malloc(nBytes);
  float* C_host=(float*)malloc(nBytes);
  float* C_from_gpu=(float*)malloc(nBytes);
  initialData(A_host,nxy);
  initialData(B_host,nxy);

  //hipMalloc
  float *A_dev=NULL;
  float *B_dev=NULL;
  float *C_dev=NULL;
  CHECK(hipMalloc((void**)&A_dev,nBytes));
  CHECK(hipMalloc((void**)&B_dev,nBytes));
  CHECK(hipMalloc((void**)&C_dev,nBytes));


  CHECK(hipMemcpy(A_dev,A_host,nBytes,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(B_dev,B_host,nBytes,hipMemcpyHostToDevice));

  int dimx=argc>2?atoi(argv[1]):32;
  int dimy=argc>2?atoi(argv[2]):32;

  double iStart,iElaps;
  // cpu compute
   iStart=cpuSecond();
  sumMatrix2D_CPU(A_host,B_host,C_host,nx,ny);
  iElaps=cpuSecond()-iStart;
  printf("CPU Execution Time elapsed %f sec\n",iElaps);
  //warm up
  // 2d block and 2d grid
  dim3 block_0(32,32);
  dim3 grid_0((nx-1)/block_0.x+1,(ny-1)/block_0.y+1);
  iStart=cpuSecond();
  sumMatrix<<<grid_0,block_0>>>(A_dev,B_dev,C_dev,nx,ny);
  CHECK(hipDeviceSynchronize());
  printf("Warm Up \n");

  // 2d block and 2d grid
  dim3 block(dimx,dimy);
  dim3 grid((nx-1)/block.x+1,(ny-1)/block.y+1);
  iStart=cpuSecond();
  sumMatrix<<<grid,block>>>(A_dev,B_dev,C_dev,nx,ny);
  CHECK(hipDeviceSynchronize());
  iElaps=cpuSecond()-iStart;
  printf("GPU Execution configuration<<<(%d,%d),(%d,%d)>>> Time elapsed %f sec\n",
        grid.x,grid.y,block.x,block.y,iElaps);
  CHECK(hipMemcpy(C_from_gpu,C_dev,nBytes,hipMemcpyDeviceToHost));

  checkResult(C_host,C_from_gpu,nxy);

  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);
  free(A_host);
  free(B_host);
  free(C_host);
  free(C_from_gpu);
  hipDeviceReset();
  return 0;
}
