#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"
#define N 300000
__global__ void kernel_1()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
__global__ void kernel_2()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
__global__ void kernel_3()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
__global__ void kernel_4()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
int main()
{
    int n_stream=4;
    hipStream_t *stream=(hipStream_t*)malloc(n_stream*sizeof(hipStream_t));
    for(int i=0;i<n_stream;i++)
    {
        hipStreamCreate(&stream[i]);
    }
    dim3 block(1);
    dim3 grid(1);
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for(int i=0;i<n_stream;i++)
    {
        kernel_1<<<grid,block,0,stream[i]>>>();
        kernel_2<<<grid,block,0,stream[i]>>>();
        kernel_3<<<grid,block>>>();
        kernel_4<<<grid,block,0,stream[i]>>>();
    }
    hipEventRecord(stop);
    CHECK(hipEventSynchronize(stop));
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time,start,stop);
    
    for(int i=0;i<n_stream;i++)
    {
        hipStreamDestroy(stream[i]);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(stream);
    return 0;
}