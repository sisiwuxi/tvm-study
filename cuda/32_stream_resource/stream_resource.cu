#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"
#define N 100
__global__ void kernel_1()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
__global__ void kernel_2()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
__global__ void kernel_3()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
__global__ void kernel_4()
{
    double sum=0.0;
    for(int i=0;i<N;i++)
        sum=sum+tan(0.1)*tan(0.1);
}
int main()
{
    //setenv("CUDA_DEVICE_MAX_CONNECTIONS","32",1);
    int n_stream=4;
    hipStream_t *stream=(hipStream_t*)malloc(n_stream*sizeof(hipStream_t));
    for(int i=0;i<n_stream;i++)
    {
        hipStreamCreate(&stream[i]);
    }
    dim3 block(16,32);
    dim3 grid(32);
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for(int i=0;i<n_stream;i++)
    {
        kernel_1<<<grid,block,0,stream[i]>>>();
        kernel_2<<<grid,block,0,stream[i]>>>();
        kernel_3<<<grid,block,0,stream[i]>>>();
        kernel_4<<<grid,block,0,stream[i]>>>();
    }
    hipEventRecord(stop);
    CHECK(hipEventSynchronize(stop));
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time,start,stop);
    printf("elapsed time:%f ms\n",elapsed_time);

    for(int i=0;i<n_stream;i++)
    {
        hipStreamDestroy(stream[i]);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(stream);
    return 0;
}
