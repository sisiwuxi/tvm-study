#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"


void sumArrays(float * a,float * b,float * res,int offset,const int size)
{

    for(int i=0,k=offset;k<size;i++,k++)
    {
        res[i]=a[k]+b[k];
    }

}
__global__ void sumArraysGPU(float*a,float*b,float*res,int offset,int n)
{
  //int i=threadIdx.x;
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int k=i+offset;
  if(k<n)
    res[i]=a[k]+b[k];
}
int main(int argc,char **argv)
{
  int dev = 0;
  hipSetDevice(dev);

  int nElem=1<<18;
  int offset=0;
  if(argc>=2)
    offset=atoi(argv[1]);
  printf("Vector size:%d\n",nElem);
  int nByte=sizeof(float)*nElem;
  float *a_h=(float*)malloc(nByte);
  float *b_h=(float*)malloc(nByte);
  float *res_h=(float*)malloc(nByte);
  float *res_from_gpu_h=(float*)malloc(nByte);
  memset(res_h,0,nByte);
  memset(res_from_gpu_h,0,nByte);

  float *a_d,*b_d,*res_d;
  CHECK(hipMalloc((float**)&a_d,nByte));
  CHECK(hipMalloc((float**)&b_d,nByte));
  CHECK(hipMalloc((float**)&res_d,nByte));
  CHECK(hipMemset(res_d,0,nByte));
  initialData(a_h,nElem);
  initialData(b_h,nElem);

  CHECK(hipMemcpy(a_d,a_h,nByte,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(b_d,b_h,nByte,hipMemcpyHostToDevice));

  dim3 block(1024);
  dim3 grid(nElem/block.x);
  double iStart,iElaps;
  iStart=cpuSecond();
  sumArraysGPU<<<grid,block>>>(a_d,b_d,res_d,offset,nElem);
  hipDeviceSynchronize();
  iElaps=cpuSecond()-iStart;
  CHECK(hipMemcpy(res_from_gpu_h,res_d,nByte,hipMemcpyDeviceToHost));
  printf("Execution configuration<<<%d,%d>>> Time elapsed %f sec --offset:%d \n",grid.x,block.x,iElaps,offset);


  sumArrays(a_h,b_h,res_h,offset,nElem);

  checkResult(res_h,res_from_gpu_h,nElem);
  hipFree(a_d);
  hipFree(b_d);
  hipFree(res_d);

  free(a_h);
  free(b_h);
  free(res_h);
  free(res_from_gpu_h);

  return 0;
}
