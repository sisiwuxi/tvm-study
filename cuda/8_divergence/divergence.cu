#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "freshman.h"
__global__ void warmup(float *c)
{
	int tid = blockIdx.x* blockDim.x + threadIdx.x;
	float a = 0.0;
	float b = 0.0;
	
	if ((tid/warpSize) % 2 == 0)
	{
		a = 100.0f;	
	}
	else
	{
		b = 200.0f;
	}
	//printf("%d %d %f \n",tid,warpSize,a+b);
	c[tid] = a + b;
}
__global__ void mathKernel1(float *c)
{
	int tid = blockIdx.x* blockDim.x + threadIdx.x;
	
	float a = 0.0;
	float b = 0.0;
	if (tid % 2 == 0)
	{
		a = 100.0f;
	}
	else
	{
		b = 200.0f;
	}
	c[tid] = a + b;
}

__global__ void mathKernel2(float *c)
{
	int tid = blockIdx.x* blockDim.x + threadIdx.x;
	float a = 0.0;
	float b = 0.0;
	if ((tid/warpSize) % 2 == 0)
	{
		a = 100.0f;
	}
	else
	{
		b = 200.0f;
	}
	c[tid] = a + b;
}
__global__ void mathKernel3(float *c)
{
	int tid = blockIdx.x* blockDim.x + threadIdx.x;
	float a = 0.0;
	float b = 0.0;
	bool ipred = (tid % 2 == 0);
	if (ipred)
	{
		a = 100.0f;
	}
	else
	{
		b = 200.0f;
	}
	c[tid] = a + b;
}

int main(int argc, char **argv)
{
	int dev = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	printf("%s using Device %d: %s\n", argv[0], dev, deviceProp.name);

	//set up data size
	int size = 64;
	int blocksize = 64;
	if (argc > 1) blocksize = atoi(argv[1]);
	if (argc > 2) size = atoi(argv[2]);
	printf("Data size %d ", size);

	//set up execution configuration
	dim3 block(blocksize,1);
	dim3 grid((size - 1) / block.x + 1,1);
	printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

	//allocate gpu memory
	float * C_dev;
	size_t nBytes = size * sizeof(float);
	float * C_host=(float*)malloc(nBytes);
	hipMalloc((float**)&C_dev, nBytes);
	
	//run a warmup kernel to remove overhead
	double iStart, iElaps;
	hipDeviceSynchronize();
	iStart = cpuSecond();
	warmup<<<grid,block>>> (C_dev);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;	
	printf("warmup	  <<<%d,%d>>>elapsed %lf sec \n", grid.x, block.x, iElaps);
	
	//run kernel 1
	iStart = cpuSecond();
	mathKernel1<<<grid,block>>> (C_dev);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	printf("mathKernel1<<<%4d,%4d>>>elapsed %lf sec \n", grid.x, block.x, iElaps);
	hipMemcpy(C_host,C_dev,nBytes,hipMemcpyDeviceToHost);
	//for(int i=0;i<size;i++)
	//{
	//	printf("%f ",C_host[i]);	
	//}

	//run kernel 2
	iStart = cpuSecond();
	mathKernel2<<<grid,block>>> (C_dev);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	printf("mathKernel2<<<%4d,%4d>>>elapsed %lf sec \n", grid.x, block.x, iElaps);

	//run kernel 3
	iStart = cpuSecond();
	mathKernel3<<<grid, block>>> (C_dev);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	printf("mathKernel3<<<%4d,%4d>>>elapsed %lf sec \n", grid.x, block.x, iElaps);

	hipFree(C_dev);
	free(C_host);
	hipDeviceReset();
	return EXIT_SUCCESS;
}
